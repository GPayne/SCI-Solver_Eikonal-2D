#include "hip/hip_runtime.h"

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//          modified to use only 7 floats for triMem
//1. #define TRIMEMLENGTH   7
//2. in FIMCuda and run_neighbor_check, add initilize old at the begining of iteration
//3. in FIMCuda and run_neighbor_check, s_triMem[tx*TRIMEMLENGTH + 3 + C] = TC after each iteration instead of s_triMem[tx*TRIMEMLENGTH + 6 + C] = TC
//4. in FIMCuda and run_neighbor_check, in the reconcile step, there should be no +3 in fetching the location of triMem
//
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#include "meshFIM.h"
#include "Vec.h"
#include <math.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "CUDADefines.h"

#include <time.h>


/////declaration for cuda kernels///////////////////////////
extern __global__ void run_reduction(int *con, int *blockCon,int* ActiveList, int nActiveBlock, int* blockSizes);
extern __global__ void FIMCuda(float* d_triMem,float* d_triMemOut, int* d_vertMem, int* d_vertMemOutside, float* d_edgeMem0,float* d_edgeMem1,float* d_edgeMem2,float* d_speed, int* d_BlockSizes, int* d_con, int* ActiveList, int nActiveBlock,int maxNumTotalFaces, int maxNumVert,/*int nIter, */float m_StopDistance);
extern __global__ void run_check_neighbor(float* d_triMem,float* d_triMemOut, int* d_vertMem,int* d_vertMemOutside,float* d_edgeMem0,float* d_edgeMem1,float* d_edgeMem2, float* d_speed, int* d_BlockSizes, int* d_con,int* d_ActiveList, int numOldActive ,int maxNumTotalFaces, int maxNumVert,int nTotalActive, int m_StopDistance);







#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}


#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	return true;
}

#endif

/////////////////////////////////////////////////////////////////////////////

void meshFIM::GraphPartition_METIS(char* partfilename, int numBlock)  //read a metis result .mesh.npart.N file and store into PartitionLabel
{
	int numVert = m_meshPtr->vertices.size();
	//m_PartitionLabel = new int[numVert];
	m_PartitionLabel.resize(numVert);
	FILE* partFile = fopen(partfilename, "r+");
	if(partFile == NULL)
	{
		printf("NO part file found!!!!\n");
		exit(1);
	}


	int tmp;

	for(int i = 0; i < numVert; i++)
	{
		fscanf(partFile, "%d", &m_PartitionLabel[i]);
		

	}

		 srand( (unsigned)time( NULL ) );




	m_BlockSizes.resize(numBlock);

	 //m_BlockSizes = new int[numBlock];
	 for(int i =0; i<numBlock; i++)
		 m_BlockSizes[i] = 0;

	float r,g,b;

	

	vector< Color > colors;
	colors.resize(numBlock);
	for(int i = 0; i< numBlock; i++)
	{
		r = rand()/(double)RAND_MAX; 
		g = rand()/(double)RAND_MAX; 
		b = rand()/(double)RAND_MAX;
		colors[i] = Color(r,g,b);
	}
	m_meshPtr->colors.resize(numVert);
	m_PartitionVerts.resize(numBlock);

	for(int i = 0; i<numVert; i++)
	{
		m_PartitionVerts[m_PartitionLabel[i]].push_back(i);

		 
		m_BlockSizes[m_PartitionLabel[i]]++;
		m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

	}

	m_maxNumVert = 0;

	for(int i = 0 ; i < numBlock; i++)
	{

		m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
	}





}

void meshFIM::GraphPartition_METIS2(int& numBlock, int maxNumBlockVerts)   //create .mesh file from trimesh faces and call partnmesh.exe program to partition and create intermediate mesh.npart.N file and then read this file
{
	
	 FILE * outf;

	 outf = fopen("tmp.mesh", "w+");
	 if(outf == NULL)
	{
		printf("Cannot open mesh file to write!!!!\n");
		exit(1);
	}
	 int sz = m_meshPtr->faces.size();
	 fprintf(outf,"%d 1\n", sz);

	 for (int i=0;i<sz;i++)
		 fprintf(outf, "%d %d %d\n",m_meshPtr->faces[i].v[0]+1,m_meshPtr->faces[i].v[1]+1,m_meshPtr->faces[i].v[2]+1);
	 fclose(outf);

	 int numVert = m_meshPtr->vertices.size();
	 
	m_PartitionLabel.resize(numVert);
	
	char outputFileName[512];



	char cmd[512];

	if(numBlock == 0)
	{
		numBlock = numVert / maxNumBlockVerts;


		do{
			numBlock++;

			m_BlockSizes.resize(numBlock);
			for(int i=0; i< numBlock;i++)
			{
				m_BlockSizes[i] = 0;
			}

			sprintf(cmd, "partnmesh.exe tmp.mesh %d", numBlock);
			//printf("running partition command: %s\n", cmd);

			system(cmd);



			sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


			FILE* partFile = fopen(outputFileName, "r+");
			if(partFile == NULL)
			{
				printf("NO part file found!!!!\n");
				exit(1);
			}


			int tmp;

			for(int i = 0; i < numVert; i++)
			{
				fscanf(partFile, "%d", &m_PartitionLabel[i]);


			}

			for(int i = 0; i<numVert; i++)
			{


				m_BlockSizes[m_PartitionLabel[i]]++;

			}
			m_maxNumVert = 0;

			for(int i = 0 ; i < numBlock; i++)
			{

				m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
			}

			fclose(partFile);


			sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
			unlink(outputFileName);
			sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
			unlink(outputFileName);

		}while(m_maxNumVert != maxNumBlockVerts);
	}
	else
	{
		m_BlockSizes.resize(numBlock);
		for(int i=0; i< numBlock;i++)
		{
			m_BlockSizes[i] = 0;
		}

		sprintf(cmd, "partnmesh.exe tmp.mesh %d", numBlock);
		//printf("running partition command: %s\n", cmd);

		system(cmd);



		sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);


		FILE* partFile = fopen(outputFileName, "r+");
		if(partFile == NULL)
		{
			printf("NO part file found!!!!\n");
			exit(1);
		}


		int tmp;

		for(int i = 0; i < numVert; i++)
		{
			fscanf(partFile, "%d", &m_PartitionLabel[i]);


		}

		for(int i = 0; i<numVert; i++)
		{


			m_BlockSizes[m_PartitionLabel[i]]++;

		}
		m_maxNumVert = 0;

		for(int i = 0 ; i < numBlock; i++)
		{

			m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
		}

		printf("max num vert is : %d\n", m_maxNumVert);
		fclose(partFile);

		
		sprintf(outputFileName, "tmp.mesh.npart.%d", numBlock);
		unlink(outputFileName);
		sprintf(outputFileName, "tmp.mesh.epart.%d", numBlock);
		unlink(outputFileName);

	}




	srand( (unsigned)time( NULL ) );

	printf("numBlock is : %d\n", numBlock);




	

	 //m_BlockSizes = new int[numBlock];
	 //for(int i =0; i<numBlock; i++)
		// m_BlockSizes[i] = 0;

	float r,g,b;

	

	vector< Color > colors;
	colors.resize(numBlock);
	for(int i = 0; i< numBlock; i++)
	{
		r = rand()/(double)RAND_MAX; 
		g = rand()/(double)RAND_MAX; 
		b = rand()/(double)RAND_MAX;
		colors[i] = Color(r,g,b);
	}
	m_meshPtr->colors.resize(numVert);
	m_PartitionVerts.resize(numBlock);

	for(int i = 0; i<numVert; i++)
	{
		m_PartitionVerts[m_PartitionLabel[i]].push_back(i);
		m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

	}

	unlink("tmp.mesh");

	

	

	



}

void meshFIM::GraphPartition_Square(int squareLength,int squareWidth, int blockLength, int blockWidth)
{
	int numVert = m_meshPtr->vertices.size();
	//m_PartitionLabel = new int[numVert];
	m_PartitionLabel.resize(numVert);

	//int numBlockEdge = (squareSize / blockSize);
	//int numBlock = numBlockEdge * numBlockEdge;

	int numBlockLength = (squareLength / blockLength);
	int numBlockWidth  = (squareWidth / blockWidth);
	int numBlock = numBlockLength * numBlockWidth;
	


	for(int i = 0; i< squareWidth; i++)
		for(int j =0; j< squareLength; j++)
		{
			m_PartitionLabel[i*squareLength+j] = (i/blockWidth) * numBlockLength + (j/blockLength);
		}

	m_BlockSizes.resize(numBlock);

	 //m_BlockSizes = new int[numBlock];
	 for(int i =0; i<numBlock; i++)
		 m_BlockSizes[i] = 0;

	float r,g,b;

	

	vector< Color > colors;
	colors.resize(numBlock);
	for(int i = 0; i< numBlock; i++)
	{
		r = rand()/(double)RAND_MAX; 
		g = rand()/(double)RAND_MAX; 
		b = rand()/(double)RAND_MAX;
		colors[i] = Color(r,g,b);
	}
	m_meshPtr->colors.resize(numVert);
	m_PartitionVerts.resize(numBlock);

	for(int i = 0; i<numVert; i++)
	{
		m_PartitionVerts[m_PartitionLabel[i]].push_back(i);

		 
		m_BlockSizes[m_PartitionLabel[i]]++;
		m_meshPtr->colors[i] = colors[m_PartitionLabel[i]];

	}

	m_maxNumVert = 0;

	for(int i = 0 ; i < numBlock; i++)
	{

		m_maxNumVert = MAX(m_maxNumVert, m_BlockSizes[i]);
	}
	printf("final number of blocks: %d\n", numBlock);





}

void meshFIM::PartitionFaces(int numBlock)
{
		/////////////////////////////////////step 3: partition faces//////////////////////////////////////
	m_PartitionFaces.resize(numBlock);
	m_PartitionNbFaces.resize(numBlock);

	int numFaces = m_meshPtr->faces.size();
	TriMesh::Face f;
	int labelv0;
	int labelv1;
	int labelv2;
	vector<TriMesh::Face> virtualfaces;
	vector<int> virtualFaceCnt;

	virtualFaceCnt.resize(numBlock);
	m_PartitionVirtualFaces.resize(numBlock);

	for(int i = 0; i< numBlock; i++)
		virtualFaceCnt[i] = 0;

	m_BlockNeighbor.resize(numBlock);

	for(int i = 0; i < numFaces; i++)
	{
		f = m_meshPtr->faces[i];
		int vfCnt = m_meshPtr->faceVirtualFaces[i].size();

		for(int k = 0 ; k < 3; k++)
		{
			if(!m_meshPtr->IsNonObtuse(f[k], f))
			{
				virtualFaceCnt[m_PartitionLabel[f[k]]] += vfCnt;
				m_PartitionVirtualFaces[m_PartitionLabel[f[k]]].insert(m_PartitionVirtualFaces[m_PartitionLabel[f[k]]].end(), m_meshPtr->faceVirtualFaces[i].begin(), m_meshPtr->faceVirtualFaces[i].end());
			}

		}





		labelv0 = m_PartitionLabel[f[0]];
		labelv1 = m_PartitionLabel[f[1]];
		labelv2 = m_PartitionLabel[f[2]];

		if(labelv0 == labelv1 && labelv1 == labelv2)
		{
			m_PartitionFaces[labelv0].push_back(i);
			//virtualFaceCnt[labelv0] += vfCnt;
		}
		else if(labelv0 == labelv1 && labelv1 != labelv2)
		{
			m_PartitionNbFaces[labelv0].push_back(i);
			m_PartitionNbFaces[labelv2].push_back(i);

			m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
			m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);


			//virtualFaceCnt[labelv0] += vfCnt;
			//virtualFaceCnt[labelv2] += vfCnt;



		}
		else if(labelv0 != labelv1 && labelv1 == labelv2)
		{
			m_PartitionNbFaces[labelv0].push_back(i);
			m_PartitionNbFaces[labelv2].push_back(i);

			m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
			m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);

			//virtualFaceCnt[labelv0] += vfCnt;
			//virtualFaceCnt[labelv2] += vfCnt;


		}

		else if(labelv0 == labelv2 && labelv1 != labelv2)
		{
			m_PartitionNbFaces[labelv0].push_back(i);
			m_PartitionNbFaces[labelv1].push_back(i);

			m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv1);
			m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv0);

			//virtualFaceCnt[labelv0] += vfCnt;
			//virtualFaceCnt[labelv1] += vfCnt;

		}

		else      //all different
		{
			m_PartitionNbFaces[labelv0].push_back(i);
			m_PartitionNbFaces[labelv1].push_back(i);
			m_PartitionNbFaces[labelv2].push_back(i);

			m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv2);
			m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv0);
			m_BlockNeighbor[labelv0].insert(m_BlockNeighbor[labelv0].end(), labelv1);
			m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv0);
			m_BlockNeighbor[labelv1].insert(m_BlockNeighbor[labelv1].end(), labelv2);
			m_BlockNeighbor[labelv2].insert(m_BlockNeighbor[labelv2].end(), labelv1);

			//virtualFaceCnt[labelv0] += vfCnt;
			//virtualFaceCnt[labelv1] += vfCnt;
			//virtualFaceCnt[labelv2] += vfCnt;


		}

	}

	vector<int> PartitionToltalFaces;
	PartitionToltalFaces.resize(numBlock);
	m_maxNumTotalFaces = 0;
	for(int j = 0; j < numBlock; j++)
	{


		PartitionToltalFaces[j] = m_PartitionFaces[j].size() + m_PartitionNbFaces[j].size() + virtualFaceCnt[j];
		m_maxNumTotalFaces = MAX(PartitionToltalFaces[j],m_maxNumTotalFaces );
	}
}




void meshFIM::GenerateData(int numBlock)
{

	int numVert = m_meshPtr->vertices.size();
	int numFaces=m_meshPtr->faces.size();

	

	

	if(!InitCUDA()) {
		exit(1);
	}

	float       *d_Vertices = 0;
	float	    *d_vertT  = 0;
	//d_Face      *d_Faces = 0;
	index       *d_ActiveList= 0;
	//int         *d_VertLabel = 0;
	int         *d_BlockLabel = 0;
	int         *d_Neighbors;
	//d_Face      *d_NonObtuseNeighborFaces;
	int        *d_con;

	int*   d_con_forComputaion;
	int        *d_blockCon;
	int         *d_blockSizes;
	float       *d_triMem;
	float      *d_edgeMem0;
	float      *d_edgeMem1;
	float      *d_edgeMem2;
	float*      d_speed;
	float      *d_triMemOut;
	float       *d_triMem_forComputation;
	int         *d_vertMem;

	int         *d_BlockSizes;
	int*      d_out;



	float       *h_Vertices = 0;
	float	    *h_vertT  = 0;
	//d_Face      *h_Faces = 0;
	index       *h_ActiveList= 0;    //list of active blocks
	//int         *h_VertLabel = 0;    //vertices active or not
	int         *h_BlockLabel = 0;   //block active or not
	int         *h_Neighbors;
	//d_Face      *h_NonObtuseNeighborFaces;
	float       *h_triMem;
	float      *h_edgeMem0;
	float      *h_edgeMem1;
	float      *h_edgeMem2;
	float*      h_speed;
	int         *h_vertMem;
	int         *h_blockCon;

	int         *h_BlockSizes;

	


	


	


	/////////////////////////////malloc cpu memories///////////////////////////
	h_BlockLabel = (int*) malloc(sizeof(int) * numBlock);
	//h_Neighbors = (int*) malloc(sizeof(int) * numVert * MAXNUMNEIGHBOR);
	//h_NonObtuseNeighborFaces = (d_Face*)malloc(sizeof(d_Face) * numVert * MAXNUMNEIGHBOR);

	h_edgeMem0 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
	h_edgeMem1 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
	h_edgeMem2 = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);
	h_speed    = (float*)malloc(sizeof(float)  * m_maxNumTotalFaces * numBlock);


	h_triMem = (float*)malloc(sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock);
	h_vertMem = (int*)malloc(sizeof(int) * VERTMEMLENGTH * m_maxNumVert * numBlock);
	h_BlockSizes = (int*)malloc(sizeof(int) * numBlock);

	h_blockCon = (int*)malloc(sizeof(int) * numBlock);



	

	/////////////////////////malloc gpu memories//////////////////////////////

	CUDA_SAFE_CALL( hipMalloc((void**) &d_con, sizeof(int) * numBlock * REDUCTIONSHARESIZE));

	CUDA_SAFE_CALL( hipMalloc((void**) &d_con_forComputaion, sizeof(int) * numBlock * REDUCTIONSHARESIZE));

	CUDA_SAFE_CALL( hipMalloc((void**) &d_blockCon,  sizeof(int) * numBlock));

	CUDA_SAFE_CALL( hipMalloc((void**) &d_triMem,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_triMemOut,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_edgeMem0,  sizeof(float)  * m_maxNumTotalFaces * numBlock));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_edgeMem1,  sizeof(float)  * m_maxNumTotalFaces * numBlock));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_edgeMem2,  sizeof(float)  * m_maxNumTotalFaces * numBlock));

	CUDA_SAFE_CALL( hipMalloc((void**) &d_speed,  sizeof(float)  * m_maxNumTotalFaces * numBlock));


	//CUDA_SAFE_CALL( hipMalloc((void**) &d_triMem_forComputation,  sizeof(float) * TRIMEMLENGTH * m_maxNumTotalFaces * numBlock));
	CUDA_SAFE_CALL( hipMalloc((void**) &d_vertMem, sizeof(int) * VERTMEMLENGTH * m_maxNumVert * numBlock));

	CUDA_SAFE_CALL( hipMalloc((void**) &d_BlockSizes, sizeof(int) * numBlock));
	//CUDA_SAFE_CALL( hipMalloc((void**) &d_out, sizeof(int) * VERTMEMLENGTH));










	/////////////////initialize cpu memories//////////////////////////////
	


	vector< vector<int> > blockVertMapping;
	blockVertMapping.resize(numVert);     //for each vertex, store the addresses where it appears in the global triMem array.


	for( int i = 0; i <  numBlock; i++)
	{
		int blockIdx = i * m_maxNumTotalFaces * TRIMEMLENGTH;
		int numPF = m_PartitionFaces[i].size();
		for(int j = 0; j< numPF; j++)
		{

			// printf("%d %f\n",m_PartitionFaces[i][j], m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0]);
			//h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0];
			//h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[1];
			//h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[2];
			
			h_edgeMem0[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[0];
			h_edgeMem1[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[1];
			h_edgeMem2[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionFaces[i][j]].edgeLens[2];


			h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
			h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM; 
			h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM; 

			h_speed[i * m_maxNumTotalFaces + j]  =  m_meshPtr->faces[m_PartitionFaces[i][j]].speedInv;



			blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
			blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
			blockVertMapping[m_meshPtr->faces[m_PartitionFaces[i][j]][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);
		}

	}

   for( int i = 0; i <  numBlock; i++)
   {

	   h_blockCon[i] = 1;

	   h_BlockLabel[i] = m_BlockLabel[i];
	   h_BlockSizes[i] = m_BlockSizes[i];
	   int blockIdx = i * m_maxNumTotalFaces * TRIMEMLENGTH;
	   int vertIdx =  i * VERTMEMLENGTH * m_maxNumVert;
	   
	   int tmpVert[3];

	   int numPF = m_PartitionFaces[i].size();
	   int numPNF = m_PartitionNbFaces[i].size();
	   int numPVF = m_PartitionVirtualFaces[i].size();

	   int k = 0;
	   int l = 0;

	   for(int j = numPF; j< m_maxNumTotalFaces; j++)
	   {
		   
		   
		 
		   if( j < numPF + numPNF)
		   {
			   h_edgeMem0[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[0];
			   h_edgeMem1[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[1];
			   h_edgeMem2[i * m_maxNumTotalFaces + j]= m_meshPtr->faces[m_PartitionNbFaces[i][k]].edgeLens[2];


			   h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM; 
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM; 
			   h_speed[i * m_maxNumTotalFaces + j] = m_meshPtr->faces[m_PartitionNbFaces[i][k]].speedInv;


			   blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
			   blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
			   blockVertMapping[m_meshPtr->faces[m_PartitionNbFaces[i][k]][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);
			   


			   k++;
			   
		   }

		   
		   else if (j < numPF + numPNF + numPVF)
		   {
			   h_edgeMem0[i * m_maxNumTotalFaces + j]= m_PartitionVirtualFaces[i][l].edgeLens[0];
			   h_edgeMem1[i * m_maxNumTotalFaces + j]= m_PartitionVirtualFaces[i][l].edgeLens[1];
			   h_edgeMem2[i * m_maxNumTotalFaces + j] = m_PartitionVirtualFaces[i][l].edgeLens[2];


			   h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM; 
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM; 
			   h_speed[i * m_maxNumTotalFaces + j]  =m_PartitionVirtualFaces[i][l].speedInv;


			   
			   blockVertMapping[m_PartitionVirtualFaces[i][l][0]].push_back(blockIdx + j*TRIMEMLENGTH + 0);
			   blockVertMapping[m_PartitionVirtualFaces[i][l][1]].push_back(blockIdx + j*TRIMEMLENGTH + 1);
			   blockVertMapping[m_PartitionVirtualFaces[i][l][2]].push_back(blockIdx + j*TRIMEMLENGTH + 2);

			   l++;


		   }
		   else
		   {
			  
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 0] = LARGENUM;
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 1] = LARGENUM;
			   h_triMem[blockIdx + j*TRIMEMLENGTH + 2] = LARGENUM;
			   //h_triMem[blockIdx + j*TRIMEMLENGTH + 3] = LARGENUM;
			   //h_triMem[blockIdx + j*TRIMEMLENGTH + 4] = LARGENUM;
			   //h_triMem[blockIdx + j*TRIMEMLENGTH + 5] = LARGENUM;
			   //h_triMem[blockIdx + j*TRIMEMLENGTH + 6] = LARGENUM;

		   }
	   }


	   
	   

   }






   m_maxNumVertMapping = 0;
   for(int i =0; i < numVert; i++)
   {
	   int blockIndex = m_PartitionLabel[i];
	   int tmp = blockVertMapping[i][0];
	   int maxi = (blockIndex+1) * m_maxNumTotalFaces * TRIMEMLENGTH;
	   int mini = blockIndex * m_maxNumTotalFaces * TRIMEMLENGTH;
	   if(  ( tmp< mini) || (tmp >= maxi) )
	   {
		   for(int j =0; j < blockVertMapping[i].size(); j++)
			   if(blockVertMapping[i][j] >= mini && blockVertMapping[i][j] < maxi )
			   {
				   int swaptmp = tmp;
				   blockVertMapping[i][0] = blockVertMapping[i][j];
				   blockVertMapping[i][j] = swaptmp;
				   break;

			   }
	   }
	   m_maxNumVertMapping = MAX(m_maxNumVertMapping, blockVertMapping[i].size());
   }


   for(int i =0; i < numVert; i++)
   {
	   int blockIndex = m_PartitionLabel[i];
	   int tmp = blockVertMapping[i][0];
	   int maxi = (blockIndex+1) * m_maxNumTotalFaces * TRIMEMLENGTH;
	   int mini = blockIndex * m_maxNumTotalFaces * TRIMEMLENGTH;
	   if(  ( tmp< mini) || (tmp >= maxi) )
	   {
		   printf("beyond");
	   }
   }



     vector< vector<int> > blockVertMappingInside;
   vector< vector<int> > blockVertMappingOutside;

   blockVertMappingInside.resize(numVert);
   blockVertMappingOutside.resize(numVert);

   for(int i = 0; i< numBlock; i++)
   {
	   int triIdx =  i * TRIMEMLENGTH * m_maxNumTotalFaces;

	   for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
	   {

		   vector<int> tmp = blockVertMapping[m_PartitionVerts[i][m]];


		   for(int n = 0; n < tmp.size(); n++)
		   {
			   if( tmp[n] >= triIdx + 0  && tmp[n] < triIdx + m_maxNumTotalFaces*TRIMEMLENGTH)
				   blockVertMappingInside[m_PartitionVerts[i][m]].push_back(tmp[n]);
			   else
			   {
				   blockVertMappingOutside[m_PartitionVerts[i][m]].push_back(tmp[n]);

			   }



		   }

	   }
   }

   int maxVertMappingInside = 0;
   int maxVertMappingOutside = 0;
   for(int i =0; i< numVert; i++)
   {
		maxVertMappingInside = MAX(maxVertMappingInside, (blockVertMappingInside[i].size()));
		maxVertMappingOutside = MAX(maxVertMappingInside, (blockVertMappingOutside[i].size()));
   }

   printf("maxVertMappingInside is: %d\n",maxVertMappingInside);
    printf("maxVertMappingOutside is: %d\n",maxVertMappingOutside);


   for(int i = 0; i< numBlock; i++)
   {
	   int vertIdx =  i * VERTMEMLENGTH * m_maxNumVert;
	   
	   for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
	   {

		   int tmpsize = blockVertMappingInside[m_PartitionVerts[i][m]].size();

		   int n = 0;
		   for(; n < tmpsize; n++)
			   h_vertMem[vertIdx + m*VERTMEMLENGTH + n] = blockVertMappingInside[m_PartitionVerts[i][m]][n];
		   for(;n<VERTMEMLENGTH; n++)
			 
			  h_vertMem[vertIdx + m*VERTMEMLENGTH + n] = -1 + i*m_maxNumTotalFaces*TRIMEMLENGTH;

	   }

	   for(int m = m_PartitionVerts[i].size() * VERTMEMLENGTH; m < m_maxNumVert * VERTMEMLENGTH; m++)
	   {
		   //h_vertMem[vertIdx + m] = -1;
		   h_vertMem[vertIdx + m] = -1 + i*m_maxNumTotalFaces*TRIMEMLENGTH;
	   }
   }


   int* h_vertMemOutside = (int*)malloc(m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE * sizeof(int));
   int* d_vertMemOutside;
   CUDA_SAFE_CALL( hipMalloc((void**) &d_vertMemOutside, m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE * sizeof(int) ) );

   for(int i = 0; i< numBlock; i++)
   {
	   int vertIdx =  i * VERTMEMLENGTHOUTSIDE * m_maxNumVert;
	   
	   for(int m  = 0; m < m_PartitionVerts[i].size(); m++)
	   {

		   int tmpsize = blockVertMappingOutside[m_PartitionVerts[i][m]].size();

		   int n = 0;
		   for(; n < tmpsize; n++)
			   h_vertMemOutside[vertIdx + m*VERTMEMLENGTHOUTSIDE + n] = blockVertMappingOutside[m_PartitionVerts[i][m]][n];
		   for(;n<VERTMEMLENGTHOUTSIDE; n++)
			   h_vertMemOutside[vertIdx + m*VERTMEMLENGTHOUTSIDE + n] = -1;

	   }

	   for(int m = m_PartitionVerts[i].size() * VERTMEMLENGTHOUTSIDE; m < m_maxNumVert * VERTMEMLENGTHOUTSIDE; m++)
	   {
		   h_vertMemOutside[vertIdx + m] = -1;
	   }
   }


   
   



   






	h_ActiveList = (int*)malloc(sizeof(int)*numBlock);
	CUDA_SAFE_CALL( hipMalloc((void**) &d_ActiveList, sizeof(int) * numBlock));



	


	


	
	






	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	
	
	float oldT1 , newT1, oldT2, newT2;
	index tmpIndex1, tmpIndex2;
	vector<int>  nb; 
//	int i;
//	int SMsize;
	int GridSize;

	
	int numActive;

	//for(int currentVert = 0; currentVert < 1/*numVert*/; currentVert++)
	//{

		for( int i = 0; i <  numBlock; i++)
		{

			h_blockCon[i] = 1;

			h_BlockLabel[i] = m_BlockLabel[i];
			h_BlockSizes[i] = m_BlockSizes[i];
		}

//////////////////////////////////initialize the seed points for h_triMem////////////////////////////////////

		for(int i = 0; i< m_SeedPoints.size(); i++)
		{
			int seed = m_SeedPoints[i];
			int seedBelongToBlock = m_PartitionLabel[seed];
			h_blockCon[seedBelongToBlock] = 0;
			// int blockIdx = seedBelongToBlock * m_maxNumTotalFaces * TRIMEMLENGTH;
			for(int j = 0; j < blockVertMapping[seed].size(); j++)
			{
				h_triMem[blockVertMapping[seed][j]] = 0.0;

			}


		}


		/////////////copy triMem and verMem to a vector just for debugging/////////////////
   vector<float> vec_triMem;
   vector<int>   vec_vertMem;
   vector<int>   vec_vertMemOutside;

   vec_triMem.resize(TRIMEMLENGTH * m_maxNumTotalFaces * numBlock);
   vec_vertMem.resize(VERTMEMLENGTH * m_maxNumVert * numBlock);
    vec_vertMemOutside.resize(VERTMEMLENGTHOUTSIDE * m_maxNumVert * numBlock);
   for(int i =0; i < TRIMEMLENGTH * m_maxNumTotalFaces * numBlock; i++)
	   vec_triMem[i] = h_triMem[i];

   for(int i = 0; i< VERTMEMLENGTH * m_maxNumVert * numBlock; i++)
	   vec_vertMem[i] = h_vertMem[i];


   for(int i = 0; i< VERTMEMLENGTHOUTSIDE * m_maxNumVert * numBlock; i++)
	   vec_vertMemOutside[i] = h_vertMemOutside[i];
   ////////////////////////////////////////////////////////////////////////////

		CUDA_SAFE_CALL( hipMemcpy( d_triMem,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));

		numActive =m_ActiveBlocks.size(); 


		set<int>::iterator activeiter = m_ActiveBlocks.begin();
		for(int i =0; activeiter !=  m_ActiveBlocks.end(); activeiter++)
			h_ActiveList[i++] = *activeiter;



		unsigned int timer = 0;
	  unsigned int timerTotal = 0;
	  unsigned int timerTotalplusCopy = 0;
	  CUT_SAFE_CALL( cutCreateTimer( &timer));

	   CUT_SAFE_CALL( cutCreateTimer( &timerTotalplusCopy));
	CUT_SAFE_CALL( cutStartTimer( timerTotalplusCopy));
	

		//////////////////copy to gpu memories///////////////////////////////

      CUDA_SAFE_CALL( hipMemcpy( d_triMem,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL( hipMemcpy( d_triMemOut,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
	   CUDA_SAFE_CALL( hipMemcpy( d_edgeMem0,h_edgeMem0, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
	   CUDA_SAFE_CALL( hipMemcpy( d_edgeMem1,h_edgeMem1, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
	   CUDA_SAFE_CALL( hipMemcpy( d_edgeMem2,h_edgeMem2, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));

	    CUDA_SAFE_CALL( hipMemcpy( d_speed,h_speed, sizeof(float) * m_maxNumTotalFaces * numBlock , hipMemcpyHostToDevice));
	  //CUDA_SAFE_CALL( hipMemcpy( d_triMem_forComputation,h_triMem, sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH, hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL( hipMemcpy( d_vertMem,h_vertMem, sizeof(int) * m_maxNumVert * numBlock * VERTMEMLENGTH, hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL( hipMemcpy( d_vertMemOutside,h_vertMemOutside, sizeof(int) * m_maxNumVert * numBlock * VERTMEMLENGTHOUTSIDE, hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL( hipMemcpy( d_BlockSizes,h_BlockSizes, sizeof(int) * numBlock, hipMemcpyHostToDevice));
	  CUDA_SAFE_CALL( hipMemcpy( d_blockCon,h_blockCon, sizeof(int) * numBlock, hipMemcpyHostToDevice));


	  printf("max number of triangles per block: %d\n", m_maxNumTotalFaces);
	  int nTotalIter = 0;
	 // int nIter = 7;

	  


	  CUT_SAFE_CALL( cutCreateTimer( &timerTotal));
	CUT_SAFE_CALL( cutStartTimer( timerTotal));
	 


	 int totalIterationNumber = 0;


		while ( numActive > 0)
		{
			

			///////////////////////////step 1: run solver //////////////////////////////////////////////////////////////////

			nTotalIter++;

			totalIterationNumber += numActive;
			//printf("number of active block: %d\n", numActive);

			dim3 dimGrid(numActive, 1);
			dim3 dimBlock(m_maxNumTotalFaces, 1);


			CUDA_SAFE_CALL( hipMemcpy( d_ActiveList,h_ActiveList,sizeof(int) * numBlock, hipMemcpyHostToDevice));

			FIMCuda<<<dimGrid, dimBlock, m_maxNumTotalFaces*TRIMEMLENGTH*sizeof(float)+m_maxNumVert*VERTMEMLENGTH*sizeof(short)>>>( d_triMem,d_triMemOut, d_vertMem,d_vertMemOutside,d_edgeMem0,d_edgeMem1,d_edgeMem2, d_speed, d_BlockSizes, d_con,d_ActiveList, numActive,m_maxNumTotalFaces, m_maxNumVert, m_StopDistance);
			CUT_CHECK_ERROR("Kernel execution failed");

			//CUDA_SAFE_CALL( hipMemcpy(h_triMem, d_triMem,sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH , hipMemcpyDeviceToHost) );
			//vec_triMem.resize(m_maxNumTotalFaces * numBlock * 3);
			//float maxVertT = 0;
			//for(int i = 0 ; i <  m_maxNumTotalFaces * numBlock; i++)
			//{

			//	
			//	vec_triMem[3*i + 0] =  h_triMem[i*TRIMEMLENGTH + 3];
			//	vec_triMem[3*i + 1] =  h_triMem[i*TRIMEMLENGTH + 4];
			//	vec_triMem[3*i + 2] =  h_triMem[i*TRIMEMLENGTH + 5];

			//	if(h_triMem[i*TRIMEMLENGTH + 3] >= LARGENUM)
			//		vec_triMem[3*i + 0] = -2;
			//	if(h_triMem[i*TRIMEMLENGTH + 4] >= LARGENUM)
			//		vec_triMem[3*i + 1] = -2;
			//	if(h_triMem[i*TRIMEMLENGTH + 5] >= LARGENUM)
			//		vec_triMem[3*i + 2] = -2;


			//	maxVertT = MAX(maxVertT,MAX(vec_triMem[3*i + 2] , MAX(vec_triMem[3*i + 1] , vec_triMem[3*i + 0])));
			//}

			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("FIMCuda Processing time: %f (ms)\n", cutGetTimerValue( timer));



			//////////////////////step 2: reduction////////////////////////////////////////////////


			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));

			//CUT_SAFE_CALL( cutStartTimer( timer));

			dimBlock = dim3(REDUCTIONSHARESIZE / 2 , 1);
			run_reduction<<<dimGrid, dimBlock/*, sizeof(int)*m_maxNumVert*/>>>(d_con, d_blockCon,d_ActiveList, numActive, d_BlockSizes);
			CUT_CHECK_ERROR("Kernel execution failed");

			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("run reduction Processing time: %f (ms)\n", cutGetTimerValue( timer));




			//////////////////////////////////////////////////////////////////
			// 3. check neighbor tiles of converged tile 
			// Add any active block of neighbor of converged block is inserted
			// to the list

	
			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));

			//CUT_SAFE_CALL( cutStartTimer( timer));


			CUDA_SAFE_CALL( hipMemcpy(h_blockCon, d_blockCon, numBlock*sizeof(int), hipMemcpyDeviceToHost) );

			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("compute neighbor copy Processing time: %f (ms)\n", cutGetTimerValue( timer));

			int nOldActiveBlock = numActive;

			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));

			//CUT_SAFE_CALL( cutStartTimer( timer));

			//vector<int> tmpActiveBlock;

			for(uint i=0; i<nOldActiveBlock; i++)
			{
				// check neighbors of current active tile
				uint currBlkIdx = h_ActiveList[i];

				if(h_blockCon[currBlkIdx]) // not active : converged
				{
					//h_BlockLabel[currBlkIdx] == FARP;
					set<int> nb = m_BlockNeighbor[currBlkIdx];

					set<int>::iterator iter;
					for( iter = nb.begin(); iter != nb.end() ; iter++)
					{
						int currIdx = *iter;

						if(h_BlockLabel[currIdx] == FARP) 
						{
							h_BlockLabel[currIdx] = ACTIVE;
							h_ActiveList[numActive++] = currIdx;
							//m_ActiveBlocks.insert(m_ActiveBlocks.end(), currIdx);
						}
					}
				}
/*				else
					h_ActiveList[numActive++] = currBlkIdx;*/   // if active block is not convergent, add it to active list and computer again next iter. a bug here: if the acitve block happen to be convengent at next iteration, there will be no active block after check_neighbor.

			}



			


			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("compute neighbor Processing time: %f (ms)\n", cutGetTimerValue( timer));


			//////////////////////////////////////////////////////////////////
			// 4. run solver only once for neighbor blocks of converged block
			// current active list contains active blocks and neighbor blocks of
			// any converged blocks
			//


			CUDA_SAFE_CALL( hipMemcpy(d_ActiveList, h_ActiveList, numActive*sizeof(int), hipMemcpyHostToDevice) );

			


			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));

			//CUT_SAFE_CALL( cutStartTimer( timer));

			dimGrid = dim3(numActive, 1);
			dimBlock = dim3(m_maxNumTotalFaces, 1);

			
			run_check_neighbor<<< dimGrid, dimBlock, m_maxNumTotalFaces*TRIMEMLENGTH*sizeof(float)+m_maxNumVert*VERTMEMLENGTH*sizeof(short)>>>(d_triMemOut, d_triMem,d_vertMem,d_vertMemOutside, d_edgeMem0,d_edgeMem1,d_edgeMem2,d_speed , d_BlockSizes, d_con,d_ActiveList, nOldActiveBlock ,m_maxNumTotalFaces, m_maxNumVert,numActive, m_StopDistance);
			CUT_CHECK_ERROR("Kernel execution failed");


			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("check neighbor Processing time: %f (ms)\n", cutGetTimerValue( timer));




			//////////////////////////////////////////////////////////////////
			// 5. reduction 


			

			dimGrid = dim3(numActive, 1);
			dimBlock = dim3(REDUCTIONSHARESIZE / 2 , 1);

			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));
			//CUT_SAFE_CALL( cutStartTimer( timer));

			run_reduction<<<dimGrid, dimBlock/*, sizeof(int)*m_maxNumVert*/>>>(d_con, d_blockCon,d_ActiveList,numActive, d_BlockSizes);
			CUT_CHECK_ERROR("Kernel execution failed");


			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("reduction2 Processing time: %f (ms)\n", cutGetTimerValue( timer));



			//////////////////////////////////////////////////////////////////
			// 6. update active list
			// read back active volume from the device and add 
			// active block to active list on the host memory


			//timer = 0;
			//CUT_SAFE_CALL( cutCreateTimer( &timer));

			//CUT_SAFE_CALL( cutStartTimer( timer));


			numActive = 0;
			//m_ActiveBlocks.clear();

			CUDA_SAFE_CALL( hipMemcpy(h_blockCon, d_blockCon, numBlock*sizeof(int), hipMemcpyDeviceToHost) );
			for(uint i=0; i<numBlock; i++)
			{
				if(!h_blockCon[i]) // false : activate block (not converged)
				{
					h_BlockLabel[i] = ACTIVE;
					h_ActiveList[numActive++] = i;	
					//m_ActiveBlocks.insert(m_ActiveBlocks.end(), i);
					//printf("Block %d added\n", i);
				}
				else h_BlockLabel[i] = FARP;
			}

			//CUDA_SAFE_CALL( hipDeviceSynchronize() );
			//CUT_SAFE_CALL( cutStopTimer( timer));
			//printf("Compute active Processing time: %f (ms)\n", cutGetTimerValue( timer));


		}
	//}

		CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//endtime = clock();
	//double duration = (double)(endtime - starttime) *1000 / CLOCKS_PER_SEC;

	//printf("Computing time : %.10lf ms\n",duration);



	CUT_SAFE_CALL( cutStopTimer( timerTotal));



		//timer = 0;
		//	CUT_SAFE_CALL( cutCreateTimer( &timer));

		//	CUT_SAFE_CALL( cutStartTimer( timer));
	CUDA_SAFE_CALL( hipMemcpy(h_triMem, d_triMem,sizeof(float) * m_maxNumTotalFaces * numBlock * TRIMEMLENGTH , hipMemcpyDeviceToHost) );

	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	CUT_SAFE_CALL( cutStopTimer( timerTotalplusCopy));

	//CUDA_SAFE_CALL( hipDeviceSynchronize() );
	//		CUT_SAFE_CALL( cutStopTimer( timer));
	//		printf("copy trimem to host Processing time: %f (ms)\n", cutGetTimerValue( timer));




	

	
	CUT_CHECK_ERROR("Kernel execution failed\n");

	
	printf("Total Processing time: %f (ms)\n", cutGetTimerValue( timerTotal));
	printf("Total Processing time and copy time: %f (ms)\n", cutGetTimerValue( timerTotalplusCopy));
	CUT_SAFE_CALL( cutDeleteTimer( timerTotal));
	CUT_SAFE_CALL( cutDeleteTimer( timer));
	CUT_SAFE_CALL( cutDeleteTimer( timerTotalplusCopy));

	printf("The iteration number: %d\n", nTotalIter );
	printf("The total iteration number: %d\n", totalIterationNumber );
	printf("The total localsolver calls per vertex: %f\n", totalIterationNumber*m_maxNumTotalFaces*(NITER+1)*3.0 / (float)numVert);

	vec_triMem.resize(m_maxNumTotalFaces * numBlock * 3);
	float maxVertT = 0;
	for(int i = 0 ; i <  m_maxNumTotalFaces * numBlock; i++)
	{

		
		vec_triMem[3*i + 0] =  h_triMem[i*TRIMEMLENGTH + 0];
		vec_triMem[3*i + 1] =  h_triMem[i*TRIMEMLENGTH + 1];
		vec_triMem[3*i + 2] =  h_triMem[i*TRIMEMLENGTH + 2];

		if(h_triMem[i*TRIMEMLENGTH + 0] >= LARGENUM)
			vec_triMem[3*i + 0] = -2;
		if(h_triMem[i*TRIMEMLENGTH + 1] >= LARGENUM)
			vec_triMem[3*i + 1] = -2;
		if(h_triMem[i*TRIMEMLENGTH + 2] >= LARGENUM)
			vec_triMem[3*i + 2] = -2;


		maxVertT = MAX(maxVertT,MAX(vec_triMem[3*i + 2] , MAX(vec_triMem[3*i + 1] , vec_triMem[3*i + 0])));
		
	}

	int vertIndex = 0;

	for(int i =0; i < numVert; i++)
	{
		
		m_meshPtr->vertT[i] =  h_triMem[blockVertMapping[i][0]];
		if(m_meshPtr->vertT[i] == maxVertT)
			vertIndex = i;


	}


	printf("The maximun vertT is: %f, the vert index is: %d \n", maxVertT,vertIndex );
	//printf("The vertT 259 is: %f\n", m_meshPtr->vertT[259] );


	//
	//printf("%s\n", h_vertT);

	//CUDA_SAFE_CALL( hipFree(d_Vertices));

	//CUDA_SAFE_CALL( hipFree(d_vertT));
	//CUDA_SAFE_CALL( hipFree(d_Faces));
	//CUDA_SAFE_CALL( hipFree(d_VertLabel));
	CUDA_SAFE_CALL( hipFree(d_ActiveList));
	CUDA_SAFE_CALL( hipFree(d_triMem));
	CUDA_SAFE_CALL( hipFree(d_vertMem));
	CUDA_SAFE_CALL( hipFree(d_edgeMem0));
	CUDA_SAFE_CALL( hipFree(d_edgeMem1));
	CUDA_SAFE_CALL( hipFree(d_edgeMem2));

	CUDA_SAFE_CALL( hipFree(d_speed));

	CUDA_SAFE_CALL( hipFree(d_con));

    CUDA_SAFE_CALL( hipFree(d_blockCon));

	//CUDA_SAFE_CALL( hipFree(d_Neighbors));

	//free(h_Vertices); 
	//free(h_vertT);
	//free(h_Faces);
	free(h_ActiveList);
	//free(h_VertLabel);
	free(h_edgeMem0);
	free(h_edgeMem1);
	free(h_edgeMem2);

	free(h_speed);

	free(h_triMem);
	free(h_vertMem);
	//free(h_Neighbors);
	free(h_BlockLabel);
	free(h_blockCon);
	free(h_BlockSizes);


	


}